#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"

int is_prime(unsigned long int number) {
    if (number <= 1) return 0;
    if (number <= 3) return 1;
    if (number % 2 == 0 || number % 3 == 0) return 0;
    for (unsigned long int i = 5; i * i <= number; i += 6) {
        if (number % i == 0 || number % (i + 2) == 0) return 0;
    }
    return 1;
}

__global__ void check_primes(unsigned long int *numbers, int *prime_counts, long inputArgument) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    long stride = gridDim.x * blockDim.x;
    int local_prime_count = 0;
    for (long i = idx; i < inputArgument; i += stride) {
        if (is_prime(numbers[i])) {
            local_prime_count++;
        }
    }
    atomicAdd(prime_counts, local_prime_count);
}

int main(int argc, char **argv) {
    Args ins__args;
    parseArgs(&ins__args, &argc, argv);

    long inputArgument = ins__args.arg;
    unsigned long int *numbers = (unsigned long int*)malloc(inputArgument * sizeof(unsigned long int));
    numgen(inputArgument, numbers);

    unsigned long int *d_numbers;
    int *d_prime_counts;
    hipMalloc((void**)&d_numbers, inputArgument * sizeof(unsigned long int));
    hipMalloc((void**)&d_prime_counts, sizeof(int));

    hipMemcpy(d_numbers, numbers, inputArgument * sizeof(unsigned long int), hipMemcpyHostToDevice);

    int zero = 0;
    hipMemcpy(d_prime_counts, &zero, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (inputArgument + blockSize - 1) / blockSize;

    struct timeval ins__tstart, ins__tstop;
    gettimeofday(&ins__tstart, NULL);

    check_primes<<<gridSize, blockSize>>>(d_numbers, d_prime_counts, inputArgument);
    hipDeviceSynchronize();

    gettimeofday(&ins__tstop, NULL);
    ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);

    int prime_count;
    hipMemcpy(&prime_count, d_prime_counts, sizeof(int), hipMemcpyDeviceToHost);

    printf("Total number of prime numbers: %d\n", prime_count);

    free(numbers);
    hipFree(d_numbers);
    hipFree(d_prime_counts);

    return 0;
}